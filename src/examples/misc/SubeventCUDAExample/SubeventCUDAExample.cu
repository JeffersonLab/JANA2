#include "hip/hip_runtime.h"

// Copyright 2020, Jefferson Science Associates, LLC.
// Subject to the terms in the LICENSE file found in the top-level directory.

#include <JANA/JApplication.h>
#include <JANA/JObject.h>
#include <JANA/JEventSource.h>
#include <JANA/JEventProcessor.h>
#include "JANA/Engine/JTopologyBuilder.h"
#include <JANA/Topology/JSubeventArrow.h>


struct MyInput : public JObject {
    int x;
    float y;
    int evt = 0;
    int sub = 0;

    MyInput(int x, float y, int evt, int sub) : x(x), y(y), evt(evt), sub(sub) {}
};

struct MyOutput : public JObject {
    float z;
    int evt = 0;
    int sub = 0;

    explicit MyOutput(float z, int evt, int sub) : z(z), evt(evt), sub(sub) {}
};

__global__ void myKernel(MyInput *in, MyOutput *out, int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) {
        out[id].z = in[id].x + in[id].y;
        out[id].evt = in[id].evt;
        out[id].sub = in[id].sub;
    }
}

void myKernelWrapper(const MyInput *h_in, MyOutput *h_out) {
    MyInput *d_in;
    MyOutput *d_out;
    hipMalloc((void **) &d_in, sizeof(MyInput));
    hipMalloc((void **) &d_out, sizeof(MyOutput));

    hipMemcpy(d_in, h_in, sizeof(MyInput), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, sizeof(MyOutput), hipMemcpyHostToDevice);

    myKernel<<<1, 1>>>(d_in, d_out, 1); // launch with only 1 GPU thread

    hipMemcpy(h_out, d_out, sizeof(MyOutput), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

struct MyProcessor : public JSubeventProcessor<MyInput, MyOutput> {
    MyProcessor() {
        inputTag = "";
        outputTag = "subeventted";
    }

    MyOutput *ProcessSubevent(MyInput *input) override {
        LOG << "Processing subevent " << input->evt << ":" << input->sub << LOG_END;

        // return new MyOutput(input->y + (float) input->x, input->evt, input->sub); // replace with CUDA here
        MyOutput *output = new MyOutput(0.0, -1, -1);
        LOG << "    Before CUDA, evt:sub=" << output->evt << ":" << output->sub << LOG_END;
        myKernelWrapper(input, output);
        LOG << "    After CUDA, evt:sub=" << output->evt << ":" << output->sub << LOG_END;
        return output;
    }
};


struct SimpleSource : public JEventSource {
    SimpleSource() {
        SetCallbackStyle(CallbackStyle::ExpertMode); 
    };

    Result Emit(JEvent& event) override {
        auto evt = event.GetEventNumber();
        std::vector < MyInput * > inputs;
        inputs.push_back(new MyInput(22, 3.6, evt, 0));
        inputs.push_back(new MyInput(23, 3.5, evt, 1));
        inputs.push_back(new MyInput(24, 3.4, evt, 2));
        inputs.push_back(new MyInput(25, 3.3, evt, 3));
        inputs.push_back(new MyInput(26, 3.2, evt, 4));
        event.Insert(inputs);
        LOG << "Emitting event " << event->GetEventNumber() << LOG_END;
        return Result::Success;
    }
};

struct SimpleProcessor : public JEventProcessor {

    SimpleProcessor() {
        SetCallbackStyle(CallbackStyle::ExpertMode);
    }

    void Process(const JEvent& event) {

        std::lock_guard <std::mutex> guard(m_mutex);

        auto outputs = event.Get<MyOutput>();
        // assert(outputs.size() == 4);
        // assert(outputs[0]->z == 25.6f);
        // assert(outputs[1]->z == 26.5f);
        // assert(outputs[2]->z == 27.4f);
        // assert(outputs[3]->z == 28.3f);
        LOG << " Contents of event " << event.GetEventNumber() << LOG_END;
        for (auto output: outputs) {
            LOG << " " << output->evt << ":" << output->sub << " " << output->z << LOG_END;
        }
        LOG << " DONE with contents of event " << event.GetEventNumber() << LOG_END;
    }
};


int main() {

    MyProcessor processor;
    JMailbox <std::shared_ptr<JEvent>> events_in;
    JMailbox <std::shared_ptr<JEvent>> events_out;
    JMailbox <SubeventWrapper<MyInput>> subevents_in;
    JMailbox <SubeventWrapper<MyOutput>> subevents_out;

    auto split_arrow = new JSplitArrow<MyInput, MyOutput>("split", &processor, &events_in, &subevents_in);
    auto subprocess_arrow = new JSubeventArrow<MyInput, MyOutput>("subprocess", &processor, &subevents_in, &subevents_out);
    auto merge_arrow = new JMergeArrow<MyInput, MyOutput>("merge", &processor, &subevents_out, &events_out);

    JApplication app;
    app.SetTimeoutEnabled(false);
    app.SetTicker(false);

    auto source = new SimpleSource("simpleSource");
    source->SetNEvents(10); // limit ourselves to 10 events. Note that the 'jana:nevents' param won't work
    // here because we aren't using JComponentManager to manage the EventSource

    auto topology = app.GetService<JTopologyBuilder>();
    auto source_arrow = new JEventSourceArrow("simpleSource", {source});
    source_arrow->set_input(topology->event_pool);
    source_arrow->set_output(&events_in);

    auto proc_arrow = new JEventMapArrow("simpleProcessor");
    proc_arrow->set_input(&events_out);
    proc_arrow->set_output(topology->event_pool);
    proc_arrow->add_processor(new SimpleProcessor);

    topology->arrows.push_back(source_arrow);
    topology->sources.push_back(source_arrow);
    topology->arrows.push_back(split_arrow);
    topology->arrows.push_back(subprocess_arrow);
    topology->arrows.push_back(merge_arrow);
    topology->arrows.push_back(proc_arrow);
    topology->sinks.push_back(proc_arrow);

    source_arrow->attach(split_arrow);
    split_arrow->attach(subprocess_arrow);
    subprocess_arrow->attach(merge_arrow);
    merge_arrow->attach(proc_arrow);

    app.Run(true);

}

